

/***************************************************************************************
* check_results()
*
* Compares the results of the GPU calculation with that of the Matlab calculation. 
* Compares results for each step of the algorithm, depending on the step_num parameter.
****************************************************************************************/
void check_results( int step_num ) {

	FILE* results_files[3];

	switch( step_num ) {
		case 2:	
			printf("\nCreating output for step 2 results: Calculation of derivatives\n");
			float *h_deriv_x = (float*) malloc(width*height*sizeof(float));
			float *h_deriv_y = (float*) malloc(width*height*sizeof(float));
			float *h_deriv_t = (float*) malloc(width*height*sizeof(float));

			char x_result[40]; 
			char y_result[40]; 
			char t_result[40];
			
			for( int i=0; i<3; i++) {
				sprintf( x_result, "results/step2/x_results%d.txt", i);
				sprintf( y_result, "results/step2/y_results%d.txt", i);
				sprintf( t_result, "results/step2/t_results%d.txt", i);

				results_files[0] = fopen( x_result, "w" );
				results_files[1] = fopen( y_result, "w" );
				results_files[2] = fopen( t_result, "w" );

				CUDA_SAFE_CALL( hipMemcpy2D( h_deriv_x, width*sizeof(float), deriv_x[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));
				CUDA_SAFE_CALL( hipMemcpy2D( h_deriv_y, width*sizeof(float), deriv_y[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));
				CUDA_SAFE_CALL( hipMemcpy2D( h_deriv_t, width*sizeof(float), deriv_t[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));

				//printf("x deriv(%d) value: %f\n",i,h_deriv_x[34*width+140]);
				//printf("y deriv(%d) value: %f\n",i,h_deriv_y[34*width+140]);
				//printf("t deriv(%d) value: %f\n",i,h_deriv_t[34*width+140]);
				
				for( int j=0; j<height; j++) {
					for( int k=0; k<width; k++ ) {
						fprintf( results_files[0], "%f\t", h_deriv_x[j*width + k] );
						fprintf( results_files[1], "%f\t", h_deriv_y[j*width + k] );
						fprintf( results_files[2], "%f\t", h_deriv_t[j*width + k] );
					}

					fprintf( results_files[0], "\n" );
					fprintf( results_files[1], "\n" );
					fprintf( results_files[2], "\n" );
				}

				for( int j=0; j<3; j++ ) fclose( results_files[j] ); 
			}
			printf("Derivative results files created.\n");

			free(h_deriv_x);
			free(h_deriv_y);
			free(h_deriv_t);
		break;
		case 3:
			printf("\nCreating output for step 3 results: Temporal Smoothing\n");
			float *h_smooth_t = (float*) malloc(width*height*sizeof(float));
			float *h_conv_x = (float*) malloc(width*height*sizeof(float));
			float *h_conv_y = (float*) malloc(width*height*sizeof(float));

			char t_smooth_result[40];
			char x_conv_result[40];
			char y_conv_result[40];
			
			for( int i=0; i<3; i++) {
				sprintf( t_smooth_result, "results/step3/t_smoothed%d.txt", i);
				sprintf( x_conv_result, "results/step3/x_conv%d.txt", i);
				sprintf( y_conv_result, "results/step3/y_conv%d.txt", i);

				results_files[0] = fopen( t_smooth_result, "w" );
				results_files[1] = fopen( x_conv_result, "w" );
				results_files[2] = fopen( y_conv_result, "w" );

				CUDA_SAFE_CALL( hipMemcpy2D( h_smooth_t, width*sizeof(float), smoothed[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));				
				CUDA_SAFE_CALL( hipMemcpy2D( h_conv_x, width*sizeof(float), convolved_x[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));				
				CUDA_SAFE_CALL( hipMemcpy2D( h_conv_y, width*sizeof(float), convolved_y[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));

				//printf("t smooth value: %f\n",h_smooth_t[4*width+4]);
				
				for( int j=0; j<height; j++) {
					for( int k=0; k<width; k++ ) {
						fprintf( results_files[0], "%f\t", h_smooth_t[j*width + k] );
						fprintf( results_files[1], "%f\t", h_conv_x[j*width + k] );
						fprintf( results_files[2], "%f\t", h_conv_y[j*width + k] );
					}

					fprintf( results_files[0], "\n" );
					fprintf( results_files[1], "\n" );
					fprintf( results_files[2], "\n" );
				}

				for( int j=0; j<3; j++ ) fclose( results_files[j] );
			}
			printf("Smoothing results files created.\n");

			free(h_conv_x);
			free(h_conv_y);
			free(h_smooth_t);
		break;
		case 4:
			printf("\nCreating output for step 4 results: 2nd Order Smoothing\n");
			float *h_smooth2 = (float*) malloc(width*height*sizeof(float));
			float *h_conv2_x = (float*) malloc(width*height*sizeof(float));
			float *h_conv2_y = (float*) malloc(width*height*sizeof(float));

			char smooth2_result[40];
			char x_conv2_result[40];
			char y_conv2_result[40];
			
			for( int i=0; i<6; i++) {
				sprintf( smooth2_result, "results/step4/smoothed_2nd%d.txt", i);
				sprintf( x_conv2_result, "results/step4/x_conv2nd%d.txt", i);
				sprintf( y_conv2_result, "results/step4/y_conv2nd%d.txt", i);

				results_files[0] = fopen( smooth2_result, "w" );
				results_files[1] = fopen( x_conv2_result, "w" );
				results_files[2] = fopen( y_conv2_result, "w" );

				CUDA_SAFE_CALL( hipMemcpy2D( h_smooth2, width*sizeof(float), smoothed_2nd[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));								
				CUDA_SAFE_CALL( hipMemcpy2D( h_conv2_x, width*sizeof(float), convolved2_x[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));				
				CUDA_SAFE_CALL( hipMemcpy2D( h_conv2_y, width*sizeof(float), convolved2_y[i], d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));


				//printf("smooth2 value: %f\n",h_smooth2[4*width+4]);
				
				for( int j=0; j<height; j++) {
					for( int k=0; k<width; k++ ) {
						fprintf( results_files[0], "%f\t", h_smooth2[j*width + k] );
						fprintf( results_files[1], "%f\t", h_conv2_x[j*width + k] );
						fprintf( results_files[2], "%f\t", h_conv2_y[j*width + k] );
					}

					fprintf( results_files[0], "\n" );
					fprintf( results_files[1], "\n" );
					fprintf( results_files[2], "\n" );
				}

				for( int j=0; j<3; j++ ) fclose( results_files[j] );
			}
			printf("Second order smoothing results files created.\n");

			free(h_conv2_x);
			free(h_conv2_y);
			free(h_smooth2);
		break;
		case 5:
			printf("\nCreating output for step 5 results: Velocity Calculation\n");
			float *h_vx = (float*) malloc(width*height*sizeof(float));
			float *h_vy = (float*) malloc(width*height*sizeof(float));

			char *vx_result = "results/step5/vx.txt";
			char *vy_result = "results/step5/vy.txt";
			
			results_files[0] = fopen( vx_result, "w" );
			results_files[1] = fopen( vy_result, "w" );

			CUDA_SAFE_CALL( hipMemcpy2D( h_vx, width*sizeof(float), Vx, d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));				
			CUDA_SAFE_CALL( hipMemcpy2D( h_vy, width*sizeof(float), Vy, d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));

			//printf("Vx value: %f\n",h_vx[34*width+140]);
			
			for( int j=0; j<height; j++) {
				for( int k=0; k<width; k++ ) {
					fprintf( results_files[0], "%f\t", h_vx[j*width + k] );
					fprintf( results_files[1], "%f\t", h_vy[j*width + k] );
				}

				fprintf( results_files[0], "\n" );
				fprintf( results_files[1], "\n" );
			}

			for( int j=0; j<2; j++ ) fclose( results_files[j] );

			printf("Velocity calculation results files created.\n");

			free(h_vx);
			free(h_vy);
		break;
		case 6:
			printf("\nCreating output for step 6 results: Smoothed Velocity\n");
			float *h_vx_smooth = (float*) malloc(width*height*sizeof(float));
			float *h_vy_smooth = (float*) malloc(width*height*sizeof(float));

			char *vx_sm_result = "results/step6/vx_sm.txt";
			char *vy_sm_result = "results/step6/vy_sm.txt";
			
			results_files[0] = fopen( vx_sm_result, "w" );
			results_files[1] = fopen( vy_sm_result, "w" );

			CUDA_SAFE_CALL( hipMemcpy2D( h_vx_smooth, width*sizeof(float), Vx_y, d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));				
			CUDA_SAFE_CALL( hipMemcpy2D( h_vy_smooth, width*sizeof(float), Vy_y, d_pitch, width*sizeof(float), height, hipMemcpyDeviceToHost));

			//printf("smoothed Vx value: %f\n",h_vx_smooth[34*width+140]);
			//printf("smoothed Vy value: %f\n",h_vy_smooth[34*width+140]);
			
			for( int j=0; j<height; j++) {
				for( int k=0; k<width; k++ ) {
					fprintf( results_files[0], "%f\t", h_vx_smooth[j*width + k] );
					fprintf( results_files[1], "%f\t", h_vy_smooth[j*width + k] );
				}

				fprintf( results_files[0], "\n" );
				fprintf( results_files[1], "\n" );
			}

			for( int j=0; j<2; j++ ) fclose( results_files[j] );

			printf("Velocity calculation results files created.\n");

			free(h_vx_smooth);
			free(h_vy_smooth);
		break;
		default:
			printf("Results not verified - nothing to verify.\n");
	}
}


