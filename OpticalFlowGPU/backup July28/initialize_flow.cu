

void init_flow() {

	//************** DERIVATIVES ***********************
	width_in_bytes = width*sizeof(float);
	
	//allocate device memory for image sequence
	for(int i=0; i<NUM_IMAGES; i++) {
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &d_img[i], &d_pitch, width_in_bytes, height) );
		CUDA_SAFE_CALL( hipMemcpy2D( d_img[i], d_pitch, sequence[i], width_in_bytes, width_in_bytes, height, hipMemcpyHostToDevice));
	}

	//allocate device memory for results
	for( int i=0; i<3; i++ ) {
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &deriv_x[i], &d_pitch, width_in_bytes, height) );
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &deriv_y[i], &d_pitch, width_in_bytes, height) );
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &deriv_t[i], &d_pitch, width_in_bytes, height) );
	}

	if( !PADDED ) d_pitch = width_in_bytes;

	//configure block and grid dimensions for grad/derivative calculation
	xgrid1 = dim3((width+XGRAD_TILE_WIDTH-1)/XGRAD_TILE_WIDTH, height);
	xblock1 = FILTER_RAD_ALIGNED+XGRAD_TILE_WIDTH+FILTER1_RAD;

	ygrid1 = dim3((width+YGRAD_TILE_WIDTH-1)/YGRAD_TILE_WIDTH, (height+YGRAD_TILE_HEIGHT-1)/YGRAD_TILE_HEIGHT);
	yblock1 = dim3(YGRAD_TILE_WIDTH, YGRAD_TILE_HEIGHT + 2*FILTER1_RAD);

	tgrid1 = dim3((width+TGRAD_TILE_WIDTH-1)/TGRAD_TILE_WIDTH, (height+TGRAD_TILE_HEIGHT-1)/TGRAD_TILE_HEIGHT);
	tblock1 = dim3(TGRAD_TILE_WIDTH,TGRAD_TILE_HEIGHT);

	pitch1 = d_pitch/sizeof(float);


	
	//************** TMP SMOOTHING ***********************
	//allocate device memory for results
	for( int i=0; i<3; i++ ) {
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &smoothed[i], &d_pitch, width_in_bytes, height) );
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &convolved_x[i], &d_pitch, width_in_bytes, height) );
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &convolved_y[i], &d_pitch, width_in_bytes, height) );
	}

	if( !PADDED ) d_pitch = width_in_bytes;

	tgrid2 = dim3((width+TSMOOTH_TILE_WIDTH-1)/TSMOOTH_TILE_WIDTH, (height+TSMOOTH_TILE_HEIGHT-1)/TSMOOTH_TILE_HEIGHT);
	tblock2 = dim3(TSMOOTH_TILE_WIDTH,TSMOOTH_TILE_HEIGHT);
	
	xgrid2 = dim3((width+XCONV_TILE_WIDTH-1)/XCONV_TILE_WIDTH, height);
	xblock2 = FILTER_RAD_ALIGNED+XCONV_TILE_WIDTH+FILTER3_RAD;

	ygrid2 = dim3((width+YCONV_TILE_WIDTH-1)/YCONV_TILE_WIDTH, (height+YCONV_TILE_HEIGHT-1)/YCONV_TILE_HEIGHT);
	yblock2 = dim3(YCONV_TILE_WIDTH, YCONV_TILE_HEIGHT + 2*FILTER3_RAD);

	pitch2 = d_pitch/sizeof(float);


	//************** 2ND ORDER SMOOTHING *****************
	for( int i=0; i<6; i++ ) {
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &smoothed_2nd[i], &d_pitch, width_in_bytes, height) );
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &convolved2_x[i], &d_pitch, width_in_bytes, height) );
		CUDA_SAFE_CALL( hipMallocPitch( (void**) &convolved2_y[i], &d_pitch, width_in_bytes, height) );
	}
	
	grid3 = dim3((width+SMOOTH2_TILE_WIDTH-1)/SMOOTH2_TILE_WIDTH, (height+SMOOTH2_TILE_HEIGHT-1)/SMOOTH2_TILE_HEIGHT);
	block3 = dim3(SMOOTH2_TILE_WIDTH,SMOOTH2_TILE_HEIGHT);

	xgrid3 = dim3((width+XCONV2_TILE_WIDTH-1)/XCONV2_TILE_WIDTH, height);
	xblock3 = FILTER_RAD_ALIGNED+XCONV2_TILE_WIDTH+FILTER4_RAD;

	ygrid3 = dim3((width+YCONV2_TILE_WIDTH-1)/YCONV2_TILE_WIDTH, (height+YCONV2_TILE_HEIGHT-1)/YCONV2_TILE_HEIGHT);
	yblock3 = dim3(YCONV2_TILE_WIDTH, YCONV2_TILE_HEIGHT + 2*FILTER4_RAD);

	xygrid = dim3((width+XYCONV_TILE_WIDTH-1)/XYCONV_TILE_WIDTH, (height+XYCONV_TILE_HEIGHT-1)/XYCONV_TILE_HEIGHT);
	xyblock = dim3(FILTER_RAD_ALIGNED+XYCONV_TILE_WIDTH+FILTER4_RAD, XYCONV_TILE_HEIGHT + 2*FILTER4_RAD);

	pitch3 = d_pitch/sizeof(float);


	//************** VELOCITY CALCULATION ****************
	CUDA_SAFE_CALL( hipMallocPitch( (void**) &Vx, &d_pitch, width_in_bytes, height) );
	CUDA_SAFE_CALL( hipMallocPitch( (void**) &Vy, &d_pitch, width_in_bytes, height) );

	grid4 = dim3((width+VELOCITY_TILE_WIDTH-1)/VELOCITY_TILE_WIDTH, (height+VELOCITY_TILE_HEIGHT-1)/VELOCITY_TILE_HEIGHT);
	block4 = dim3(VELOCITY_TILE_WIDTH,VELOCITY_TILE_HEIGHT);

	pitch4 = d_pitch/sizeof(float);


	//************** VELOCITY SMOOTHING *****************
	CUDA_SAFE_CALL( hipMallocPitch( (void**) &Vx_x, &d_pitch, width_in_bytes, height) );
	CUDA_SAFE_CALL( hipMallocPitch( (void**) &Vy_x, &d_pitch, width_in_bytes, height) );
	CUDA_SAFE_CALL( hipMallocPitch( (void**) &Vx_y, &d_pitch, width_in_bytes, height) );
	CUDA_SAFE_CALL( hipMallocPitch( (void**) &Vy_y, &d_pitch, width_in_bytes, height) );

	xgrid5 = dim3((width+XSMOOTHV_TILE_WIDTH-1)/XSMOOTHV_TILE_WIDTH, height);
	xblock5 = FILTER_RAD_ALIGNED+XSMOOTHV_TILE_WIDTH+FILTER5_RAD;

	ygrid5 = dim3((width+YSMOOTHV_TILE_WIDTH-1)/YSMOOTHV_TILE_WIDTH, (height+YSMOOTHV_TILE_HEIGHT-1)/YSMOOTHV_TILE_HEIGHT);
	yblock5 = dim3(YSMOOTHV_TILE_WIDTH, YSMOOTHV_TILE_HEIGHT + 2*FILTER5_RAD);

	pitch5 = d_pitch/sizeof(float);


	//GET RESULTS
	CUDA_SAFE_CALL( hipHostMalloc( (void**) &vx_final, width*height*sizeof(float)));
	CUDA_SAFE_CALL( hipHostMalloc( (void**) &vy_final, width*height*sizeof(float)));
	//vx_final = (float*) malloc(width*height*sizeof(float));
	//vy_final = (float*) malloc(width*height*sizeof(float));



}
