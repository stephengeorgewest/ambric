#include "hip/hip_runtime.h"

/* 
 * Device code.
 */

#ifndef _SMOOTH_KERNEL_H_
#define _SMOOTH_KERNEL_H_

/**********************************************************************
* smooth_t()
*
* This kernel temporally smooths the derivative images using [2 4 2]
* mask.
*
***********************************************************************/
__global__ void 
smooth_t(float *d_Result, float *frame0, float *frame1, float *frame2, int width, int height, int pitch) {

	__shared__ float data[TSMOOTH_TILE_HEIGHT*TSMOOTH_TILE_WIDTH*(2*FILTER2_RAD+1)];

	const int columnStart = MUL(blockIdx.x, TSMOOTH_TILE_WIDTH) + threadIdx.x;
	const int rowStart = MUL(blockIdx.y, TSMOOTH_TILE_HEIGHT) + threadIdx.y;

    	const int columnClamped = min(columnStart, width-1);
	const int rowClamped = min(rowStart, height-1);

	const int tile_size = MUL(TSMOOTH_TILE_WIDTH,TSMOOTH_TILE_HEIGHT);

	int gmem = MUL(rowClamped, pitch) + columnClamped;

	// Cache tile in the shared memoy array
	int smem = MUL(threadIdx.y, TSMOOTH_TILE_WIDTH) + threadIdx.x;

	data[smem] = frame0[gmem];
	data[smem + tile_size] = frame1[gmem];
	data[smem + 2*tile_size] = frame2[gmem];

	__syncthreads();

    	// Filter output pixels per thread
	if (columnStart < width && rowStart < height) {
		float sum=0;

		if(!(columnStart<2 || columnStart>=width-2 || rowStart<2 || rowStart>=height-2))
			sum = 2*data[smem] + 4*data[smem+tile_size] + 2*data[smem+2*tile_size];

		#if( MATCH_ROGER != 0 ) 
		{
			sum = (float)rintf(sum/2.0f);
			if( sum > 255 ) sum = 255;
			if( sum < -256 ) sum = -256;
		}
		#endif

		d_Result[gmem] = sum;
	}

}

/**********************************************************************
* convolve_x()
*
* This kernel calculates the convolution for an image in the x direction 
* using a [3 3 4 3 3] filter.  Non-tiled - works best if width is
* multiple of 16 or 32.  Only used when width <= 512, the max number
* of threads.
*
***********************************************************************/
extern __shared__ float sharedConvMem[];

__global__ void 
convolve_x(float *d_Result, float *d_Data, int width, int height, int pitch) {

	int I = threadIdx.x;
	int B = blockIdx.x;
	//__shared__ float sequence[IMAGE_WIDTH];
	float* sequence = sharedConvMem;

	sequence[I] = d_Data[B*width + I];

	__syncthreads();

	float resultX = 0.0;

	if( !(I < 2 || I >= width-2) && !(B < 2 || B >= height-2) ) {
		//calculate gradient in the x direction
		resultX =  3*sequence[I-2] + 3*sequence[I-1] + 4*sequence[I] + 3*sequence[I+1] + 3*sequence[I+2];
		
		#if( MATCH_ROGER ) 
		{
			resultX = (float)rintf(resultX/4.0f);
			if( resultX > 1023 ) resultX = 1023;
			if( resultX < -1024 ) resultX = -1024;
		}
		#endif
	}

	d_Result[B*width + I] = resultX; 
}

/**********************************************************************
* convolve_x_t()
*
* This kernel calculates the convolution for an image in the x direction 
* using a [3 3 4 3 3] filter.  Tiled - works for any image size.  A
* bit slower than non-tiled if width < 512 and is a multiple of 16.
*
***********************************************************************/
__global__ void 
convolve_x_t(float *d_Result, float *d_Data, int width, int height, int pitch) {

	__shared__ float data[XCONV_TILE_WIDTH + 2*FILTER3_RAD];

	//Current tile and apron limits, relative to row start
	const int         tileStart = IMUL(blockIdx.x, XCONV_TILE_WIDTH);
	const int           tileEnd = tileStart + XCONV_TILE_WIDTH - 1;
	const int        apronStart = tileStart - FILTER3_RAD;
	const int          apronEnd = tileEnd   + FILTER3_RAD;

	//Clamp tile and apron limits by image borders
	const int    tileEndClamped = min(tileEnd, width - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, width - 1);

	//Row start index in d_Data[]
	const int          rowStart = IMUL(blockIdx.y, pitch);

	const int apronStartAligned = tileStart - FILTER_RAD_ALIGNED;

	const int loadPos = apronStartAligned + threadIdx.x;
	//Set the entire data cache contents
	//Load global memory values, if indices are within the image borders,
	//or initialize with zeroes otherwise
	if(loadPos >= apronStart) { //first 16 threads not used to load?
		const int smemPos = loadPos - apronStart;

		data[smemPos] =  ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ? d_Data[rowStart + loadPos] : 0;
	}

	__syncthreads();

	const int writePos = tileStart + threadIdx.x;
	//Assuming width and XGRAD_TILE_WIDTH are multiples of half-warp size,
	//rowStart + tileStart is also a multiple of half-warp size,
	//thus having proper alignment for coalesced d_Result[] write.
	if(writePos <= tileEndClamped) {
		const int smemPos = writePos - apronStart;
		float sum = 0;

		if(!(writePos<FILTER3_RAD || writePos>=width-FILTER3_RAD || blockIdx.y<FILTER3_RAD || blockIdx.y>=height-FILTER3_RAD))
			sum =  3*data[smemPos-2] + 3*data[smemPos-1] + 4*data[smemPos] + 3*data[smemPos+1] + 3*data[smemPos+2];

		#if( MATCH_ROGER ) 
		{
			sum = (float)rintf(sum/4.0f);
			if( sum > 1023 ) sum = 1023;
			if( sum < -1024 ) sum = -1024;
		}
		#endif

		d_Result[rowStart + writePos] = sum;
	}
}


/**********************************************************************
* convolve_y()
*
* This kernel calculates the convolution for an image in the y direction 
* using a [3 3 4 3 3] filter.
*
***********************************************************************/
__global__ void 
convolve_y(float *d_Result, float *d_Data, int image_width, int image_height, int pitch) {

	__shared__ float data[YCONV_TILE_WIDTH*(2*FILTER3_RAD+YCONV_TILE_HEIGHT)];

	//Current tile and apron limits, in rows
	const int         tileStart = IMUL(blockIdx.y, YCONV_TILE_HEIGHT);
	const int           tileEnd = tileStart + YCONV_TILE_HEIGHT - 1;
	const int        apronStart = tileStart - FILTER3_RAD;
	const int          apronEnd = tileEnd   + FILTER3_RAD;

	//Clamp tile and apron limits by image borders
	// const int    tileEndClamped = min(tileEnd, image_height - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, image_height - 1);

	//Current column index
	const int       columnStart = IMUL(blockIdx.x, YCONV_TILE_WIDTH) + threadIdx.x;

	if(columnStart < image_width) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y, YCONV_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(apronStart + threadIdx.y, pitch) + columnStart;

		//Load global memory values, if indices are within the image borders,
		//or initialize with zero otherwise
		data[smemPos] = ((apronStart + threadIdx.y >= apronStartClamped) && (apronStart + threadIdx.y <= apronEndClamped)) 
				? d_Data[gmemPos] : 0;
	}
	
	__syncthreads();

	if(columnStart < image_width && threadIdx.y < YCONV_TILE_HEIGHT && tileStart + threadIdx.y < image_height) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y + FILTER3_RAD, YCONV_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(tileStart + threadIdx.y, pitch) + columnStart;

		float sum = 0;

		if(!( ( columnStart < FILTER3_RAD ) || ( columnStart >= (image_width-FILTER3_RAD) ) || ( (tileStart + threadIdx.y) < FILTER3_RAD) || ( (tileStart + threadIdx.y) >= (image_height-FILTER3_RAD) ) ))
			sum = 3*data[smemPos-2*YCONV_TILE_WIDTH] + 3*data[smemPos-1*YCONV_TILE_WIDTH] + 4*data[smemPos] + 3*data[smemPos+1*YCONV_TILE_WIDTH] + 3*data[smemPos+2*YCONV_TILE_WIDTH];
		
		#if( MATCH_ROGER ) 
		{
			sum = (float)rintf(sum/4.0f);
			if( sum > 8191 ) sum = 8191;
			if( sum < -8192 ) sum = -8192;
		}
		#endif

		d_Result[gmemPos] = sum;
	}
}






#endif // #ifndef _SMOOTH_KERNEL_H_



