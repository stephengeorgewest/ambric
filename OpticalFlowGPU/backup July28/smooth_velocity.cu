#include "hip/hip_runtime.h"

/* 
 * Device code.
 */

#ifndef _VSMOOTH_KERNEL_H_
#define _VSMOOTH_KERNEL_H_



/**********************************************************************
* smooth_velocity_x()
*
* This kernel calculates the convolution for an image in the x direction 
* using a [1 1 1 2 1 1 1] filter.  Non-tiled - works best if width is
* multiple of 16 or 32.  Only used when width <= 512, the max number
* of threads.
*
***********************************************************************/
extern __shared__ float sharedMemSV[];

__global__ void 
smooth_velocity_x(float *d_Result, float *d_Data, int width, int height, int pitch) {

	int I = threadIdx.x;
	int B = blockIdx.x;
	float* sequence = sharedMemSV;

	sequence[I] = d_Data[B*width + I];

	__syncthreads();

	float resultX = 0.0;

	if( !(I < 3 || I >= width-3) && !(B < 3 || B >= height-3) ) {
		//calculate gradient in the x direction
		resultX =  .125*sequence[I-3] + .125*sequence[I-2] + .125*sequence[I-1] + .25*sequence[I] + .125*sequence[I+1] + .125*sequence[I+2] + .125*sequence[I+3];
	}

	d_Result[B*width + I] = resultX; 
}

/**********************************************************************
* smooth_velocity_x_t()
*
* This kernel calculates the convolution for an image in the x direction 
* using a [1 1 1 2 1 1 1] filter.  Tiled - works for any image size.  A
* bit slower than non-tiled if width < 512 and is a multiple of 16.
*
***********************************************************************/
__global__ void 
smooth_velocity_x_t(float *d_Result, float *d_Data, int width, int height, int pitch) {

	__shared__ float data[FILTER5_RAD + XSMOOTHV_TILE_WIDTH + FILTER5_RAD];

	//Current tile and apron limits, relative to row start
	const int         tileStart = IMUL(blockIdx.x, XSMOOTHV_TILE_WIDTH);
	const int           tileEnd = tileStart + XSMOOTHV_TILE_WIDTH - 1;
	const int        apronStart = tileStart - FILTER5_RAD;
	const int          apronEnd = tileEnd   + FILTER5_RAD;

	//Clamp tile and apron limits by image borders
	const int    tileEndClamped = min(tileEnd, width - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, width - 1);

	//Row start index in d_Data[]
	const int          rowStart = IMUL(blockIdx.y, pitch);

	const int apronStartAligned = tileStart - FILTER_RAD_ALIGNED;

	const int loadPos = apronStartAligned + threadIdx.x;
	//Set the entire data cache contents
	//Load global memory values, if indices are within the image borders,
	//or initialize with zeroes otherwise
	if(loadPos >= apronStart){
	const int smemPos = loadPos - apronStart;

	data[smemPos] = 
	        ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ?
	        d_Data[rowStart + loadPos] : 0;
	}

	__syncthreads();

	const int writePos = tileStart + threadIdx.x;
	//Assuming width and XSMOOTHV_TILE_WIDTH are multiples of half-warp size,
	//rowStart + tileStart is also a multiple of half-warp size,
	//thus having proper alignment for coalesced d_Result[] write.
	if(writePos <= tileEndClamped){
	const int smemPos = writePos - apronStart;
	float sum = 0;

	if(!(writePos<FILTER5_RAD || writePos>=width-FILTER5_RAD || blockIdx.y<FILTER5_RAD || blockIdx.y>=height-FILTER5_RAD))
		sum = .125*data[smemPos-3] + .125*data[smemPos-2] + .125*data[smemPos-1] + .25*data[smemPos] + .125*data[smemPos+1] + .125*data[smemPos+2] + .125*data[smemPos+3];

		d_Result[rowStart + writePos] = sum;
	}
}


/**********************************************************************
* smooth_velocity_y()
*
* This kernel calculates the convolution for an image in the y direction 
* using a [1 1 1 2 1 1 1] filter.
*
***********************************************************************/
__global__ void 
smooth_velocity_y(float *d_Result, float *d_Data, int image_width, int image_height, int pitch) {
    
	__shared__ float data[YSMOOTHV_TILE_WIDTH * (FILTER5_RAD + YSMOOTHV_TILE_HEIGHT + FILTER5_RAD)];

	//Current tile and apron limits, in rows
	const int         tileStart = IMUL(blockIdx.y, YSMOOTHV_TILE_HEIGHT);
	const int           tileEnd = tileStart + YSMOOTHV_TILE_HEIGHT - 1;
	const int        apronStart = tileStart - FILTER5_RAD;
	const int          apronEnd = tileEnd   + FILTER5_RAD;

	//Clamp tile and apron limits by image borders
	// const int    tileEndClamped = min(tileEnd, height - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, image_height - 1);

	//Current column index
	const int       columnStart = IMUL(blockIdx.x, YSMOOTHV_TILE_WIDTH) + threadIdx.x;

	if(columnStart < image_width) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y, YSMOOTHV_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(apronStart + threadIdx.y, pitch) + columnStart;

		//Load global memory values, if indices are within the image borders,
		//or initialize with zero otherwise
		data[smemPos] = ((apronStart + threadIdx.y >= apronStartClamped) && (apronStart + threadIdx.y <= apronEndClamped)) ? d_Data[gmemPos] : 0;
	}
	
	__syncthreads();

	if(columnStart < image_width && threadIdx.y < YSMOOTHV_TILE_HEIGHT && tileStart + threadIdx.y < image_height) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y + FILTER5_RAD, YSMOOTHV_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(tileStart + threadIdx.y, pitch) + columnStart;

		float sum = 0;

		if(!( ( columnStart < FILTER5_RAD ) || ( columnStart >= (image_width-FILTER5_RAD) ) || ( (tileStart + threadIdx.y) < FILTER5_RAD) || ( (tileStart + threadIdx.y) >= (image_height-FILTER5_RAD) ) ))
			sum = .125*data[smemPos-3*YSMOOTHV_TILE_WIDTH] + .125*data[smemPos-2*YSMOOTHV_TILE_WIDTH] + .125*data[smemPos-1*YSMOOTHV_TILE_WIDTH] + .25*data[smemPos] + .125*data[smemPos+1*YSMOOTHV_TILE_WIDTH] + .125*data[smemPos+2*YSMOOTHV_TILE_WIDTH] + .125*data[smemPos+3*YSMOOTHV_TILE_WIDTH];

		d_Result[gmemPos] = sum;
	}
}








#endif // #ifndef _VSMOOTH_KERNEL_H_


