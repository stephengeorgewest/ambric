#include "hip/hip_runtime.h"

/* 
 * Device code.
 */

#ifndef _SMOOTH2_KERNEL_H_
#define _SMOOTH2_KERNEL_H_

/**********************************************************************
* second_smooth()
*
* similar to outer product kernel of previous optical flow
***********************************************************************/
__global__ void 
second_smooth(float *gx, float *gy, float *gt, float *xx, float *yy, float *tt, float *xy, float *xt, float *yt, int image_width, int image_height, int pitch) {

	int idx = MUL(blockIdx.x, SMOOTH2_TILE_WIDTH) + threadIdx.x;
	int idy = MUL(blockIdx.y, SMOOTH2_TILE_HEIGHT) + threadIdx.y;
	int index = MUL(idy,pitch) + idx;

	if(idx < image_width && idy < image_height) {
		//float tmpX = DIV(gx[index],32.0f);
		//float tmpY = DIV(gy[index],32.0f);
		//float tmpT = DIV(gt[index],32.0f);

		/*xy[index] = tmpX * tmpY;
		yy[index] = tmpY * tmpY;
		yt[index] = tmpY * tmpT;
		tt[index] = tmpT * tmpT;
		xt[index] = tmpT * tmpX;
		xx[index] = tmpX * tmpX;*/

		xy[index] = gx[index] * gy[index];
		yy[index] = gy[index] * gy[index];
		yt[index] = gy[index] * gt[index];
		tt[index] = gt[index] * gt[index];
		xt[index] = gt[index] * gx[index];
		xx[index] = gx[index] * gx[index];
	}
}

/**********************************************************************
* second_smooth_match()
*
* The results of this kernel are scaled and saturated to match the 
* MATLAB results
***********************************************************************/
__global__ void 
second_smooth_match(float *gx, float *gy, float *gt, float *xx, float *yy, float *tt, float *xy, float *xt, float *yt, int image_width, int image_height, int pitch) {

	int idx = MUL(blockIdx.x, SMOOTH2_TILE_WIDTH) + threadIdx.x;
	int idy = MUL(blockIdx.y, SMOOTH2_TILE_HEIGHT) + threadIdx.y;
	int index = MUL(idy,pitch) + idx;
	float tmp[6];

	if(idx < image_width && idy < image_height) {		

		tmp[0] = (float)rintf(gx[index]*gx[index] / 1024.0f);
		tmp[1] = (float)rintf(gy[index]*gy[index] / 1024.0f);
		tmp[2] = (float)rintf(gt[index]*gt[index] / 1024.0f);
		tmp[3] = (float)rintf(gx[index]*gy[index] / 1024.0f);
		tmp[4] = (float)rintf(gy[index]*gt[index] / 1024.0f);
		tmp[5] = (float)rintf(gx[index]*gt[index] / 1024.0f);

		for( int i=0; i<6; i++ ) {
			if( tmp[i] > 8191.0f ) tmp[i] = 8191.0f;
			if( tmp[i] < -8192.0f ) tmp[i] = -8192.0f;		
		}		

		xx[index] = tmp[0];
		yy[index] = tmp[1];
		tt[index] = tmp[2];
		xy[index] = tmp[3];
		yt[index] = tmp[4];
		xt[index] = tmp[5];
	}
}


/**********************************************************************
* convolve2_x()
*
* This kernel calculates the convolution for an image in the x direction 
* using a [5 6 5] filter.  Non-tiled - works best if width is
* multiple of 16 or 32.  Only used when width <= 512, the max number
* of threads.
*
***********************************************************************/
extern __shared__ float sharedConv2Mem[];

__global__ void 
convolve2_x(float *d_Result, float *d_Data, int width, int height, int pitch) {

	int I = threadIdx.x;
	int B = blockIdx.x;
	//__shared__ float sequence[IMAGE_WIDTH];
	float* sequence = sharedConv2Mem;

	sequence[I] = d_Data[B*width + I];

	__syncthreads();

	float resultX = 0.0;

	if( !(I < 1 || I >= width-1) && !(B < 1 || B >= height-1) ) {
		//calculate gradient in the x direction
		resultX =  5*sequence[I-1] + 6*sequence[I] + 5*sequence[I+1];
		
		#if( MATCH_ROGER ) 
		{
			resultX = (float)rintf(resultX/8.0f);
			if( resultX > 16383 ) resultX = 16383;
			if( resultX < -16384 ) resultX = -16384;
		}
		#endif
	}

	d_Result[B*width + I] = resultX; 
}

/**********************************************************************
* convolve2_x_t()
*
* This kernel calculates the convolution for an image in the x direction 
* using a [5 6 5] filter.  Tiled - works for any image size.  A
* bit slower than non-tiled if width < 512 and is a multiple of 16.
*
***********************************************************************/
__global__ void 
convolve2_x_t(float *d_Result, float *d_Data, int width, int height, int pitch) {

	__shared__ float data[XCONV2_TILE_WIDTH + 2*FILTER4_RAD + 1];

	//Current tile and apron limits, relative to row start
	const int         tileStart = IMUL(blockIdx.x, XCONV2_TILE_WIDTH);
	const int           tileEnd = tileStart + XCONV2_TILE_WIDTH - 1;
	const int        apronStart = tileStart - FILTER4_RAD;
	const int          apronEnd = tileEnd   + FILTER4_RAD;

	//Clamp tile and apron limits by image borders
	const int    tileEndClamped = min(tileEnd, width - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, width - 1);

	//Row start index in d_Data[]
	const int          rowStart = IMUL(blockIdx.y, pitch);

	const int apronStartAligned = tileStart - FILTER_RAD_ALIGNED;

	const int loadPos = apronStartAligned + threadIdx.x;
	//Set the entire data cache contents
	//Load global memory values, if indices are within the image borders,
	//or initialize with zeroes otherwise
	if(loadPos >= apronStart) {
		const int smemPos = loadPos - apronStart;

		data[smemPos] =  ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ? d_Data[rowStart + loadPos] : 0;
	}

	__syncthreads();

	const int writePos = tileStart + threadIdx.x;
	//Assuming width and XGRAD_TILE_WIDTH are multiples of half-warp size,
	//rowStart + tileStart is also a multiple of half-warp size,
	//thus having proper alignment for coalesced d_Result[] write.
	if(writePos <= tileEndClamped) {
		const int smemPos = writePos - apronStart;
		float sum = 0;

		if(!(writePos<FILTER4_RAD || writePos>=width-FILTER4_RAD || blockIdx.y<FILTER4_RAD || blockIdx.y>=height-FILTER4_RAD))
			sum =  5*data[smemPos-1] + 6*data[smemPos] + 5*data[smemPos+1];

		#if( MATCH_ROGER ) 
		{
			sum = (float)rintf(sum/8.0f);
			if( sum > 16383 ) sum = 16383;
			if( sum < -16384 ) sum = -16384;
		}
		#endif

		d_Result[rowStart + writePos] = sum;
	}
}


/**********************************************************************
* convolve2_y()
*
* This kernel calculates the convolution for an image in the y direction 
* using a [5 6 5] filter.
*
***********************************************************************/
__global__ void 
convolve2_y(float *d_Result, float *d_Data, int image_width, int image_height, int pitch) {

	__shared__ float data[YCONV2_TILE_WIDTH*(2*FILTER4_RAD+1+YCONV2_TILE_HEIGHT)];

	//Current tile and apron limits, in rows
	const int         tileStart = IMUL(blockIdx.y, YCONV2_TILE_HEIGHT);
	const int           tileEnd = tileStart + YCONV2_TILE_HEIGHT - 1;
	const int        apronStart = tileStart - FILTER4_RAD;
	const int          apronEnd = tileEnd   + FILTER4_RAD;

	//Clamp tile and apron limits by image borders
	// const int    tileEndClamped = min(tileEnd, image_height - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, image_height - 1);

	//Current column index
	const int       columnStart = IMUL(blockIdx.x, YCONV2_TILE_WIDTH) + threadIdx.x;

	if(columnStart < image_width) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y, YCONV2_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(apronStart + threadIdx.y, pitch) + columnStart;

		//Load global memory values, if indices are within the image borders,
		//or initialize with zero otherwise
		data[smemPos] = ((apronStart + threadIdx.y >= apronStartClamped) && (apronStart + threadIdx.y <= apronEndClamped)) 
				? d_Data[gmemPos] : 0;
	}
	
	__syncthreads();

	if(columnStart < image_width && threadIdx.y < YCONV2_TILE_HEIGHT && tileStart + threadIdx.y < image_height) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y + FILTER4_RAD, YCONV2_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(tileStart + threadIdx.y, pitch) + columnStart;

		float sum = 0;

		if(!( ( columnStart < FILTER4_RAD ) || ( columnStart >= (image_width-FILTER4_RAD) ) || ( (tileStart + threadIdx.y) < FILTER4_RAD) || ( (tileStart + threadIdx.y) >= (image_height-FILTER4_RAD) ) ))
			sum = 5*data[smemPos-1*YCONV2_TILE_WIDTH] + 6*data[smemPos] + 5*data[smemPos+1*YCONV2_TILE_WIDTH];
		
		#if( MATCH_ROGER ) 
		{
			sum = (float)rintf(sum/8.0f); 
			if( sum > 32767 ) sum = 32767;
			if( sum < -32768 ) sum = -32768;
			
			// this scaling is done at the beginning of the velocity calculation in the matlab code.  
			sum = (float)rintf(sum/8.0f); 
		}
		#endif

		d_Result[gmemPos] = sum;
	}
}


/**********************************************************************
* convolve2_xy()
*
* This kernel calculates the convolution for an image in the y direction 
* using a 3x3 filter.
*
***********************************************************************/
extern __shared__ float sharedMemxy[];

__global__ void 
convolve2_xy(float *d_Result, float *d_Data, int image_width, int image_height) {

	int I = threadIdx.x;	//represents column
	int B = blockIdx.x;	//represents row
	
	if( B>=image_height ) {
		B=B-image_height;	  	
		I=I+320;	
	}
	
	if( B==0 || B==image_height-1 ) {
		//zero out the top and bottom rows
		d_Result[B*image_width + I] = 0;
	}
	else {	
		float* data0 = (float*)sharedMemxy;
		float* data1 = (float*)&data0[image_width];
		float* data2 = (float*)&data1[image_width];
		
		for( int i = threadIdx.x; i<image_width; i+=320 ) {		
			data0[i] = d_Data[(B-1)*image_width + i];
			data1[i] = d_Data[B*image_width + i];
			data2[i] = d_Data[(B+1)*image_width + i];
			
		}
		__syncthreads();

		for( int i = threadIdx.x; i<image_width; i+=320 ) {	
			if( i==0 || i==image_width-1 ) {		
				//zero out first and last columns
				d_Result[B*image_width + i] = 0;
			}
			else {
				float sum = 25*data0[i-1] + 30*data0[i] + 25*data0[i+1]
			    	+ 30*data1[i-1] + 36*data1[i] + 30*data1[i+1]
			    	+ 25*data2[i-1] + 30*data2[i] + 25*data2[i+1];

				d_Result[B*image_width + i] = sum;
			}
		}
	}


	/*__shared__ float data[(2*FILTER4_RAD+YCONV2_TILE_WIDTH)*(2*FILTER4_RAD+1+YCONV2_TILE_HEIGHT)];

	//Current tile and apron limits, in rows
	const int         tileStart = IMUL(blockIdx.y, YCONV2_TILE_HEIGHT);
	const int           tileEnd = tileStart + YCONV2_TILE_HEIGHT - 1;
	const int        apronStart = tileStart - FILTER4_RAD;
	const int          apronEnd = tileEnd   + FILTER4_RAD;

	//Clamp tile and apron limits by image borders
	// const int    tileEndClamped = min(tileEnd, image_height - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, image_height - 1);

	//Current column index
	const int       columnStart = IMUL(blockIdx.x, YCONV2_TILE_WIDTH) + threadIdx.x;

	if(columnStart < image_width) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y, YCONV2_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(apronStart + threadIdx.y, pitch) + columnStart;

		//Load global memory values, if indices are within the image borders,
		//or initialize with zero otherwise
		data[smemPos] = ((apronStart + threadIdx.y >= apronStartClamped) && (apronStart + threadIdx.y <= apronEndClamped)) 
				? d_Data[gmemPos] : 0;
	}
	
	__syncthreads();

	if(columnStart < image_width && threadIdx.y < YCONV2_TILE_HEIGHT && tileStart + threadIdx.y < image_height) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y + FILTER4_RAD, YCONV2_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(tileStart + threadIdx.y, pitch) + columnStart;

		float sum = 0;

		if(!( ( columnStart < FILTER4_RAD ) || ( columnStart >= (image_width-FILTER4_RAD) ) || ( (tileStart + threadIdx.y) < FILTER4_RAD) || ( (tileStart + threadIdx.y) >= (image_height-FILTER4_RAD) ) ))
			sum = 25*data[smemPos-YCONV2_TILE_WIDTH-1] + 30*data[smemPos-YCONV2_TILE_WIDTH] + 25*data[smemPos-YCONV2_TILE_WIDTH+1]
			    + 30*data[smemPos-1] + 36*data[smemPos] + 30*data[smemPos+1]
			    + 25*data[smemPos+YCONV2_TILE_WIDTH-1] + 30*data[smemPos+YCONV2_TILE_WIDTH] + 25*data[smemPos+YCONV2_TILE_WIDTH+1];

		d_Result[gmemPos] = sum;
	}
	*/
}





#endif // #ifndef _SMOOTH2_KERNEL_H_

