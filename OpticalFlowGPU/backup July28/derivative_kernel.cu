#include "hip/hip_runtime.h"

/* 
 * Device code.
 */

#ifndef _DERIV_KERNEL_H_
#define _DERIV_KERNEL_H_

#define IMUL(x,y) __mul24(x,y)
#define MUL(A,B) __mul24(A, B)
#define IDIV(x, y) ((int)((x) / (float)(y)))


/**********************************************************************
* derivative_x()
*
* This kernel calculates the derivative for an image in the x direction 
* using a [-1 8 0 -8 1] filter.  Non-tiled - works best if width is
* multiple of 16 or 32.  Only used when width <= 512, the max number
* of threads.
*
***********************************************************************/
extern __shared__ float sharedMem[];

__global__ void 
derivative_x(float *d_Result, float *d_Data, int width, int height, int pitch) {

	int I = threadIdx.x;
	int B = blockIdx.x;
	//__shared__ float sequence[IMAGE_WIDTH];
	float* sequence = sharedMem;

	sequence[I] = d_Data[B*width + I];

	__syncthreads();

	float resultX = 0.0;

	if( !(I < 2 || I >= width-2) && !(B < 2 || B >= height-2) ) {
		//calculate gradient in the x direction
		resultX =  -1*sequence[I-2] + 8*(sequence[I-1] - sequence[I+1]) + sequence[I+2];
		
		#if( MATCH_ROGER ) 
		{
			resultX = (float)rintf(resultX/8.0);
			if( resultX > 127 ) resultX = 127;
			if( resultX < -127 ) resultX = -127;
		}
		#endif
	}

	d_Result[B*width + I] = resultX; 
}

/**********************************************************************
* derivative_x_t()
*
* This kernel calculates the derivative for an image in the x direction 
* using a [-1 8 0 -8 1] filter.  Tiled - works for any image size.  A
* bit slower than non-tiled if width < 512 and is a multiple of 16.
*
***********************************************************************/
__global__ void 
derivative_x_t(float *d_Result, float *d_Data, int width, int height, int pitch) {

	__shared__ float data[XGRAD_TILE_WIDTH + 2*FILTER1_RAD];

	//Current tile and apron limits, relative to row start
	const int         tileStart = IMUL(blockIdx.x, XGRAD_TILE_WIDTH);
	const int           tileEnd = tileStart + XGRAD_TILE_WIDTH - 1;
	const int        apronStart = tileStart - FILTER1_RAD;
	const int          apronEnd = tileEnd   + FILTER1_RAD;

	//Clamp tile and apron limits by image borders
	const int    tileEndClamped = min(tileEnd, width - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, width - 1);

	//Row start index in d_Data[]
	const int          rowStart = IMUL(blockIdx.y, pitch);

	const int apronStartAligned = tileStart - FILTER_RAD_ALIGNED;

	const int loadPos = apronStartAligned + threadIdx.x;
	//Set the entire data cache contents
	//Load global memory values, if indices are within the image borders,
	//or initialize with zeroes otherwise
	if(loadPos >= apronStart) {
		const int smemPos = loadPos - apronStart;

		data[smemPos] =  ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ? d_Data[rowStart + loadPos] : 0;
	}

	__syncthreads();

	const int writePos = tileStart + threadIdx.x;
	//Assuming width and XGRAD_TILE_WIDTH are multiples of half-warp size,
	//rowStart + tileStart is also a multiple of half-warp size,
	//thus having proper alignment for coalesced d_Result[] write.
	if(writePos <= tileEndClamped) {
		const int smemPos = writePos - apronStart;
		float sum = 0;

		if(!(writePos<FILTER1_RAD || writePos>=width-FILTER1_RAD || blockIdx.y<FILTER1_RAD || blockIdx.y>=height-FILTER1_RAD))
			sum = -1 * data[smemPos-2] + 8 * data[smemPos-1] - 8 * data[smemPos+1] + data[smemPos+2];

		#if( MATCH_ROGER ) 
		{
			sum = (float)rintf(sum/8.0);
			if( sum > 127 ) sum = 127;
			if( sum < -127 ) sum = -127;
		}
		#endif

		d_Result[rowStart + writePos] = sum;
	}
}


/**********************************************************************
* derivative_y()
*
* This kernel calculates the derivative for an image in the y direction 
* using a [-1 8 0 -8 1] filter.
*
***********************************************************************/
__global__ void 
derivative_y(float *d_Result, float *d_Data, int image_width, int image_height, int pitch) {

	__shared__ float data[YGRAD_TILE_WIDTH*(2*FILTER1_RAD+YGRAD_TILE_HEIGHT)];

	//Current tile and apron limits, in rows
	const int         tileStart = IMUL(blockIdx.y, YGRAD_TILE_HEIGHT);
	const int           tileEnd = tileStart + YGRAD_TILE_HEIGHT - 1;
	const int        apronStart = tileStart - FILTER1_RAD;
	const int          apronEnd = tileEnd   + FILTER1_RAD;

	//Clamp tile and apron limits by image borders
	// const int    tileEndClamped = min(tileEnd, image_height - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, image_height - 1);

	//Current column index
	const int       columnStart = IMUL(blockIdx.x, YGRAD_TILE_WIDTH) + threadIdx.x;

	if(columnStart < image_width) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y, YGRAD_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(apronStart + threadIdx.y, pitch) + columnStart;

		//Load global memory values, if indices are within the image borders,
		//or initialize with zero otherwise
		data[smemPos] = ((apronStart + threadIdx.y >= apronStartClamped) && (apronStart + threadIdx.y <= apronEndClamped)) 
				? d_Data[gmemPos] : 0;
	}
	
	__syncthreads();

	if(columnStart < image_width && threadIdx.y < YGRAD_TILE_HEIGHT && tileStart + threadIdx.y < image_height) {
		//Shared and global memory indices for current column
		int smemPos = IMUL(threadIdx.y + FILTER1_RAD, YGRAD_TILE_WIDTH) + threadIdx.x;
		int gmemPos = IMUL(tileStart + threadIdx.y, pitch) + columnStart;

		float sum = 0;

		if(!( ( columnStart < FILTER1_RAD ) || ( columnStart >= (image_width-FILTER1_RAD) ) || ( (tileStart + threadIdx.y) < FILTER1_RAD) || ( (tileStart + threadIdx.y) >= (image_height-FILTER1_RAD) ) ))
			sum = -1*data[smemPos-2*YGRAD_TILE_WIDTH] + 8*(data[smemPos-1*YGRAD_TILE_WIDTH] - data[smemPos+1*YGRAD_TILE_WIDTH]) + data[smemPos+2*YGRAD_TILE_WIDTH];
		
		#if( MATCH_ROGER != 0 ) 
		{
			sum = (float)rintf(sum/8.0);
			if( sum > 127 ) sum = 127;
			if( sum < -127 ) sum = -127;
		}
		#endif

		d_Result[gmemPos] = sum;
	}
}




/**********************************************************************
* derivative_t()
*
* This kernel calculates the derivative for an image in the time
* direction using a [-1 8 0 -8 1] filter on 5 different images.
*
***********************************************************************/
__global__ void 
derivative_t(float *d_Result, float *img0, float *img1, float *img2, float *img3, int width, int height, int pitch) {

	__shared__ float data[TGRAD_TILE_HEIGHT*TGRAD_TILE_WIDTH*2*FILTER1_RAD];

	const int columnStart = MUL(blockIdx.x, TGRAD_TILE_WIDTH) + threadIdx.x;
	const int rowStart = MUL(blockIdx.y, TGRAD_TILE_HEIGHT) + threadIdx.y;

    	const int columnClamped = min(columnStart, width-1);
	const int rowClamped = min(rowStart, height-1);

	const int tile_size = MUL(TGRAD_TILE_WIDTH,TGRAD_TILE_HEIGHT);

	int gmem = MUL(rowClamped, pitch) + columnClamped;

	// Cache tile in the shared memoy array
	int smem = MUL(threadIdx.y, TGRAD_TILE_WIDTH) + threadIdx.x;

	data[smem] = img0[gmem];
	data[smem + tile_size] = img1[gmem];
	data[smem + 2*tile_size] = img2[gmem];
	data[smem + 3*tile_size] = img3[gmem];

	__syncthreads();

    	// Filter output pixels per thread
	if (columnStart < width && rowStart < height) {
		float sum=0;

		if(!(columnStart<2 || columnStart>=width-2 || rowStart<2 || rowStart>=height-2))
			sum = -1*data[smem] + 8*data[smem+tile_size] - 8*data[smem+2*tile_size] + data[smem+3*tile_size];

		#if( MATCH_ROGER != 0 ) 
		{
			sum = (float)rintf(sum/8.0);
			if( sum > 127 ) sum = 127;
			if( sum < -127 ) sum = -127;
		}
		#endif

		d_Result[gmem] = sum;
	}
}




#endif // #ifndef _DERIV_KERNEL_H_




