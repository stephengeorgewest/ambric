#include "hip/hip_runtime.h"

#ifndef _VELOCITY_KERNEL_H_
#define _VELOCITY_KERNEL_H_


__global__ void 
velocity_kernel(float *vx, float* vy, float *xx, float *yy, float *tt, float *xy, float *xt, float *yt, int img_width, int img_height, int pitch) {

	int idx = MUL(blockIdx.x, VELOCITY_TILE_WIDTH) + threadIdx.x;
	int idy = MUL(blockIdx.y, VELOCITY_TILE_HEIGHT) + threadIdx.y;
	int index = MUL(idy,pitch) + idx;

	if(idx < img_width && idy < img_height) {

		//shared memory is not used in this kernel, but none of these values will be accessed by other threads
		float tmp1x2 = xx[index] * yy[index]; //GxGx.*GyGy
		float tmp1x5 = xx[index] * yt[index]; //GxGx.*GyGt
		float tmp3x5 = xy[index] * yt[index]; //GxGy.*GyGt
		float tmp3x4 = xy[index] * xt[index]; //GxGy.*GxGt
		float tmp2x4 = yy[index] * xt[index]; //GyGy.*GxGt
		float tmp3x3 = xy[index] * xy[index]; //GxGy.*GxGy

		float Vx_dividend = tmp2x4 - tmp3x5;	//GyGy.*GxGt - GxGy.*GyGt
		float Vy_dividend = tmp1x5 - tmp3x4;	//GxGx.*GyGt - GxGy.*GxGt
		float divisor = tmp1x2 - tmp3x3;	//GxGx.*GyGy - GxGy.*GxGy

		float tempVx, tempVy;

		if( divisor == 0 ) {
			tempVx = 0;
			tempVy = 0;
		} else {
			tempVx = DIV(Vx_dividend, divisor);
			tempVy = DIV(Vy_dividend, divisor);
		}

		if( divisor < 0.5f ) {
			//k must be calculated, velocity recalculated - Roger recalculates velocity using the velocity calculated in the previous pixel 
			//(either to the left or up one pixel).  I am doing it using the velocity originally calculated for that pixel. 
			float k = (tt[index] - 2*xt[index]*tempVx - 2*yt[index]*tempVy + xx[index]*tempVx*tempVx + 2*xy[index]*tempVx*tempVy + yy[index]*tempVy*tempVy)/28.0f;
			
			float gxx_n = xx[index] + k;
			float gyy_n = yy[index] + k;

			tmp1x2 = gxx_n * gyy_n; 	//GxGx.*GyGy
			tmp1x5 = gxx_n * yt[index]; 	//GxGx.*GyGz
			tmp2x4 = gyy_n * xt[index]; 	//GyGy.*GxGz

			Vx_dividend = tmp2x4 - tmp3x5;	//GyGy.*GxGt - GxGy.*GyGt
			Vy_dividend = tmp1x5 - tmp3x4;	//GxGx.*GyGz - GxGy.*GxGz
			divisor = tmp1x2 - tmp3x3;	//GxGx.*GyGy - GxGy.*GxGy			

			if( divisor == 0 ) {
				tempVx = 0;
				tempVy = 0;
			} else {
				tempVx = DIV(Vx_dividend, divisor);
				tempVy = DIV(Vy_dividend, divisor);
			}
		}

		#if( MATCH_ROGER ) 
		{
			if( tempVx > 7 ) tempVx = 7;
			if( tempVx < -8 ) tempVx = -8;
			if( tempVy > 7 ) tempVy = 7;
			if( tempVy < -8 ) tempVy = -8;
		}
		#endif

		vx[index] = tempVx;
		vy[index] = tempVy;

	}

}



#endif // #ifndef _VELOCITY_KERNEL_H_
