/***************************************************************************************
* John Bodily
* Brigham Young University
* CHREC project B2
*
* Started June 12, 2008
*
* Optical flow on the GPU
****************************************************************************************/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <cstdio>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <cutil.h>

// includes, kernels
#include <flow08.h>
#include <initialize_flow.cu>
#include <derivative_kernel.cu>
#include <smoothing_kernel.cu>
#include <second_smooth_kernel.cu>
#include <velocity_kernel.cu>
#include <smooth_velocity.cu>
#include <check_results.cu>

/***************************************************************************************
* read_images()
*
* 1st step: Read in an image sequence from memory.  Currently reads an entire sequence.  
* In the future streaming functionality may be added.
****************************************************************************************/
void read_images( char* argv[] ) 
{
	char *filepath, tmppath[50];
	char *ext, *read_type;
	char num[5];
	int start_image = 0;
	total_images = 0;

	assert( strlen(argv[1]) < 15 );
	strcpy( seq_name, argv[1] );

	if( strcmp(seq_name, "yosemite") == 0 ) {
		printf( "Running yosemite sequence\n" );
		total_images = 15;
		start_image = 2;
		read_type = "rb";
		filepath = "images/raw yos sequence/yos";
		ext = ".raw";
		width = 316;
		height = 252;
	} else if( strcmp(seq_name, "flower garden") == 0 ) {
		printf( "Running flower garden sequence\n" );
		total_images = 31;
		read_type = "r";
		filepath = "images/raw flower garden/flowg";
		ext = ".raw";
		width = 352;
		height = 240;
	} 
	else {
		printf( "Unrecognized image sequence\n" );
		exit(0);
	}

	FILE* images[MAX_IMAGES];

	// open up image files
	for( int i=0; i<total_images; i++ ) {
		strcpy( tmppath, filepath );
		sprintf(num, "%d", i+start_image);
		strcat(tmppath,num);
		strcat(tmppath,ext);	
		
		images[i] = fopen(tmppath, read_type);	
		if( !images[i] ) {
			printf("ERROR: Invalid image file: %s\n", tmppath);
			exit(0);
		}	

		//printf("Image to read in: %s\n", tmppath);
	}

	// allocate host memory for the sequence of images	
	sequence = (float **) malloc( total_images * sizeof(float*) );
	for( int i=0; i<total_images; i++) 
		sequence[i] = (float *) malloc( width*height*sizeof(float)  );

	//read the images into host memory
	for( int i=0; i<total_images; i++) {
		for( int j=0; j<width*height; j++ ) 
			sequence[i][j] = (float)getc( images[i] );
	}

	//close the image files now that the info is on the heap
	for( int i=0; i<total_images; i++) 
		fclose(images[i]);
}

/***************************************************************************************
* read_random_images()
*
* 1st step: Fill image sequence with random data for performance testing
****************************************************************************************/
void read_random_images( char* argv[], int argc ) 
{
	assert( strlen(argv[1]) < 15 );
	strcpy( seq_name, argv[1] );

	if( strcmp(seq_name, "yosemite") == 0 ) {
		width = 316;
		height = 252;
	} else if( strcmp(seq_name, "flower garden") == 0 ) {
		width = 352;
		height = 240;
	} 
	else if( strcmp(seq_name, "random") == 0 ) {
		if( argc != 5 ) {
			printf("Usage:\n%s %s 2 [width] [height]\n", argv[0], seq_name);
			exit(0);
		}
		width = atoi(argv[3]);
		height = atoi(argv[4]);
	}
	else {
		printf( "Unrecognized image sequence\n" );
		exit(0);
	}

	sequence = (float **) malloc( NUM_IMAGES * sizeof(float*) );
	for( int i=0; i<NUM_IMAGES; i++) 
		sequence[i] = (float *) malloc( width*height*sizeof(float)  );

	// fill sequence with random data
	for( int i=0; i<NUM_IMAGES; i++ ) {	
		for( int j=0; j<width*height; j++ ) 
			sequence[i][j] = (float) rand()/RAND_MAX;
	}
}



/***************************************************************************************
* calculate_derivates()
*
* 2nd step in Roger's new algorithm.  Similar to gradient calculation in previous 
* algorithm, except done on 3 images rather than 1
****************************************************************************************/
void calculate_derivatives( int mid ) 
{
	for( int i=0; i<3; i++ ) {
		if( width<=512 && width%16 == 0 )
			derivative_x<<<height,width,width_in_bytes>>>(deriv_x[i], d_img[mid-1+i], width, height, pitch1); //faster if width<512
		else
			derivative_x_t<<<xgrid1,xblock1>>>(deriv_x[i], d_img[mid-1+i], width, height, pitch1); //tiled version
		CUT_CHECK_ERROR("Kernel Execution Failed");

		derivative_y<<<ygrid1,yblock1>>>(deriv_y[i], d_img[mid-1+i], width, height, pitch1);   //tiled version - faster for y
		CUT_CHECK_ERROR("Kernel Execution Failed");

		derivative_t<<<tgrid1,tblock1>>>(deriv_t[i], d_img[mid-3+i], d_img[mid-2+i], d_img[mid+i], d_img[mid+1+i], width, height, pitch1);
		CUT_CHECK_ERROR("Kernel Execution Failed");
	}

	hipDeviceSynchronize();
}



/***************************************************************************************
* temporal_smoothing()
*
* 3rd step in Roger's new algorithm.  Temporally smooth over 3 derivative result frames
* using [2 4 2] mask.  Followed by x and y convolution on result using [3 3 4 3 3].
****************************************************************************************/
void temporal_smoothing() 
{
	smooth_t<<<tgrid2,tblock2>>>(smoothed[0], deriv_x[0], deriv_x[1], deriv_x[2], width, height, pitch2);
	CUT_CHECK_ERROR("Kernel Execution Failed");

	smooth_t<<<tgrid2,tblock2>>>(smoothed[1], deriv_y[0], deriv_y[1], deriv_y[2], width, height, pitch2);
	CUT_CHECK_ERROR("Kernel Execution Failed");

	smooth_t<<<tgrid2,tblock2>>>(smoothed[2], deriv_t[0], deriv_t[1], deriv_t[2], width, height, pitch2);
	CUT_CHECK_ERROR("Kernel Execution Failed");

	hipDeviceSynchronize();

	for( int i=0; i<3; i++ ) {
		if( width<=512 && width%16 == 0 )
			convolve_x<<<height,width,width_in_bytes>>>(convolved_x[i], smoothed[i], width, height, pitch2); //faster if width<512
		else
			convolve_x_t<<<xgrid2,xblock2>>>(convolved_x[i], smoothed[i], width, height, pitch2); //tiled version
		CUT_CHECK_ERROR("Kernel Execution Failed");
		
		convolve_y<<<ygrid2,yblock2>>>(convolved_y[i], convolved_x[i], width, height, pitch2);   //tiled version - faster for y
		CUT_CHECK_ERROR("Kernel Execution Failed");

	}

	hipDeviceSynchronize();
}


/***************************************************************************************
* second_order_smoothing()
*
* 4th step in Roger's new algorithm.  Calculates outer product, then smooths each of 6
* frames using [5 6 5] mask. 
****************************************************************************************/
void second_order_smoothing() 
{
	#if( MATCH_ROGER )
		second_smooth_match<<<grid3,block3>>>(convolved_y[0], convolved_y[1], convolved_y[2], smoothed_2nd[0], smoothed_2nd[1],
						smoothed_2nd[2], smoothed_2nd[3], smoothed_2nd[4], smoothed_2nd[5], width, height, pitch3);
	#else
		second_smooth<<<grid3,block3>>>(convolved_y[0], convolved_y[1], convolved_y[2], smoothed_2nd[0], smoothed_2nd[1],
						smoothed_2nd[2], smoothed_2nd[3], smoothed_2nd[4], smoothed_2nd[5], width, height, pitch3);
	#endif

	for( int i=0; i<6; i++ ) {
		/**/if( width<=512 && width%16 == 0 )
			convolve2_x<<<height,width,width_in_bytes>>>(convolved2_x[i], smoothed_2nd[i], width, height, pitch2); //faster if width<512
		else
			convolve2_x_t<<<xgrid3,xblock3>>>(convolved2_x[i], smoothed_2nd[i], width, height, pitch2); //tiled version
		CUT_CHECK_ERROR("Kernel Execution Failed");
		
		convolve2_y<<<ygrid3,yblock3>>>(convolved2_y[i], convolved2_x[i], width, height, pitch2);   //tiled version - faster for y
		CUT_CHECK_ERROR("Kernel Execution Failed");
		
		//convolve_xy<<<height,320,width*sizeof(float)*3>>>(convolved2_y[i], smoothed_2nd[i], width, height);
		//convolve_xy_t<<<xygrid,xyblock>>>(convolved2_y[i], smoothed_2nd[i], width, height, pitch3);

	}

	hipDeviceSynchronize();
}

/***************************************************************************************
* calculate_velocity()
*
* 5th step in Roger's new algorithm.  
****************************************************************************************/
void calculate_velocity() 
{
	velocity_kernel<<<grid4,block4>>>(Vx, Vy, convolved2_y[0], convolved2_y[1], convolved2_y[2], convolved2_y[3], convolved2_y[4],
						convolved2_y[5], width, height, pitch4);
	CUT_CHECK_ERROR("Kernel Execution Failed");
	
	hipDeviceSynchronize();
}

/***************************************************************************************
* smooth_velocity()
*
* 6th step in Roger's new algorithm.  
****************************************************************************************/
void smooth_velocity() 
{
	if( width<=512 && width%16 == 0 ) {
		smooth_velocity_x<<<xgrid5,xblock5,width_in_bytes>>>(Vx_x, Vx, width, height, pitch5);
		CUT_CHECK_ERROR("Kernel Execution Failed");

		smooth_velocity_x<<<xgrid5,xblock5,width_in_bytes>>>(Vy_x, Vy, width, height, pitch5);
		CUT_CHECK_ERROR("Kernel Execution Failed");
	}
	else {
		smooth_velocity_x_t<<<xgrid5,xblock5>>>(Vx_x, Vx, width, height, pitch5);
		CUT_CHECK_ERROR("Kernel Execution Failed");

		smooth_velocity_x_t<<<xgrid5,xblock5>>>(Vy_x, Vy, width, height, pitch5);
		CUT_CHECK_ERROR("Kernel Execution Failed");
	}

	smooth_velocity_y<<<ygrid5,yblock5>>>(Vx_y, Vx_x, width, height, pitch5);
	CUT_CHECK_ERROR("Kernel Execution Failed");

	smooth_velocity_y<<<ygrid5,yblock5>>>(Vy_y, Vy_x, width, height, pitch5);
	CUT_CHECK_ERROR("Kernel Execution Failed");
	
	hipDeviceSynchronize();
}



/***************************************************************************************
* clean_up()
*
****************************************************************************************/
void clean_up() {
	//clean up memory
	free( sequence );

	for( int i=0; i<3; i++ ) {
		CUDA_SAFE_CALL( hipFree(convolved_x[i]) );
		CUDA_SAFE_CALL( hipFree(convolved_y[i]) );
		CUDA_SAFE_CALL( hipFree(smoothed[i]) );
		CUDA_SAFE_CALL( hipFree(deriv_x[i]) );
		CUDA_SAFE_CALL( hipFree(deriv_y[i]) );
		CUDA_SAFE_CALL( hipFree(deriv_t[i]) );
	}

	for( int i=0; i<6; i++ ) {
		CUDA_SAFE_CALL( hipFree(smoothed_2nd[i]) );
		CUDA_SAFE_CALL( hipFree(convolved2_x[i]) );
		CUDA_SAFE_CALL( hipFree(convolved2_y[i]) );
	}

	CUDA_SAFE_CALL( hipFree(Vx) );
	CUDA_SAFE_CALL( hipFree(Vy) );
	CUDA_SAFE_CALL( hipFree(Vx_x) );
	CUDA_SAFE_CALL( hipFree(Vy_x) );
	CUDA_SAFE_CALL( hipFree(Vx_y) );
	CUDA_SAFE_CALL( hipFree(Vy_y) );
	CUDA_SAFE_CALL(hipHostFree(vx_final));
	CUDA_SAFE_CALL(hipHostFree(vy_final));

	CUT_SAFE_CALL( cutDeleteTimer( timer));
}

/***************************************************************************************
* Main()
*
****************************************************************************************/
int main(int argc, char *argv[]) {
	if( argc < 3 || argc > 5 ) {
		printf( "Usage:"
			"\nOption 1 (timing by kernel):"
			"\n%s [yosemite|\"flower garden\"] 1"
			"\nOption 2 (timing by frame):"
			"\n%s [yosemite|\"flower garden\"|random] 2 [width] [height] "
			"\nNote: random option requires a width and height\nOption 3 (write out results to verify):"
			"\n%s [yosemite|\"flower garden\"] 3\n", argv[0], argv[0], argv[0] );
		exit(0);
	}

	CUT_DEVICE_INIT();
	empty_kernel<<<1,1>>>();	//initialize runtime libraries.  Is this still necessary?

	//timer to measure performance
	CUT_SAFE_CALL( cutCreateTimer( &timer));	

	if( atoi(argv[2]) == 1 ) {
		printf( "Running with timing by kernel, size = %dx%d\n", width, height );

		//Step 1: read images into memory and initialize GPU variables
		read_images( argv );
		init_flow();

		//Step 2
		CUT_SAFE_CALL( cutStartTimer( timer));
		calculate_derivatives( 3 ); 			// ( middle_image )
		CUT_SAFE_CALL( cutStopTimer( timer));
		float deriv_time = cutGetTimerValue(timer);
		float total_time = deriv_time;

		//Step 3
		CUT_SAFE_CALL( cutResetTimer( timer));
		CUT_SAFE_CALL( cutStartTimer( timer));
		temporal_smoothing();			
		CUT_SAFE_CALL( cutStopTimer( timer));
		float smooth_time = cutGetTimerValue(timer);	
		total_time += smooth_time;

		//Step 4
		CUT_SAFE_CALL( cutResetTimer( timer));
		CUT_SAFE_CALL( cutStartTimer( timer));
		second_order_smoothing(); 		
		CUT_SAFE_CALL( cutStopTimer( timer));
		float smooth2_time = cutGetTimerValue(timer);	
		total_time += smooth2_time;

		//Step 5
		CUT_SAFE_CALL( cutResetTimer( timer));
		CUT_SAFE_CALL( cutStartTimer( timer));
		calculate_velocity(); 		
		CUT_SAFE_CALL( cutStopTimer( timer));
		float veloc_time = cutGetTimerValue(timer);	
		total_time += veloc_time;

		//Step 6
		CUT_SAFE_CALL( cutResetTimer( timer));
		CUT_SAFE_CALL( cutStartTimer( timer));
		smooth_velocity(); 	
		CUT_SAFE_CALL( cutStopTimer( timer));
		float sm_veloc_time = cutGetTimerValue(timer);	
		total_time += sm_veloc_time;

		//get results 
		CUT_SAFE_CALL( cutResetTimer( timer));
		CUT_SAFE_CALL( cutStartTimer( timer));

		CUDA_SAFE_CALL( hipMemcpy2D( vx_final, width_in_bytes, Vx, d_pitch, width_in_bytes, height, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL( hipMemcpy2D( vy_final, width_in_bytes, Vy, d_pitch, width_in_bytes, height, hipMemcpyDeviceToHost));

		CUT_SAFE_CALL( cutStopTimer( timer));
		float veloc_cpy_time = cutGetTimerValue(timer);	
		total_time += veloc_cpy_time;


		printf("\nDerivative time = %f ms\n",deriv_time); 
		printf("Total temporal smoothing time = %f ms\n",smooth_time);
		printf("Second order smoothing time = %f ms\n",smooth2_time);  
		printf("Velocity calculation time = %f ms\n",veloc_time); 
		printf("Smooth velocity time = %f ms\n",sm_veloc_time); 
		printf("Copy velocity results back to host = %f ms\n",veloc_cpy_time); 
		printf("Total calculation time = %f ms\n",total_time); 
		printf("------------------------------------------------\n"); 

	} else if( atoi(argv[2]) == 2 ) {
		const int TRIALS = 100;
		read_random_images( argv, argc );
		//read_images( argv );
		init_flow();
		
		printf( "Running with timing by frame for %d random data frames, size = %dx%d\n", TRIALS, width, height );
		
		float* h_img;
		CUDA_SAFE_CALL( hipHostMalloc( (void**) &h_img, width*height*sizeof(float)));
		//float *h_img = (float*) malloc(width*height*sizeof(float));

		for(int i=0; i<width*height; i++)
			h_img[i] = (float) rand()/RAND_MAX;
			
		for(int i=0; i<TRIALS; i++) {

			CUT_SAFE_CALL( cutStartTimer( timer));

			float *tmp = d_img[NUM_IMAGES-1];
			for(int img=NUM_IMAGES-1; img>0; img--)
				d_img[img] = d_img[img-1];
			d_img[0] = tmp;

			CUDA_SAFE_CALL( hipMemcpy2D( d_img[0], d_pitch, h_img, width_in_bytes, width_in_bytes, height, hipMemcpyHostToDevice));

			calculate_derivatives( 3 ); 	
			temporal_smoothing();
			second_order_smoothing(); 
			calculate_velocity(); 
			smooth_velocity(); 	
			CUDA_SAFE_CALL( hipMemcpy2D( vx_final, width_in_bytes, Vx, d_pitch, width_in_bytes, height, hipMemcpyDeviceToHost));
			CUDA_SAFE_CALL( hipMemcpy2D( vy_final, width_in_bytes, Vy, d_pitch, width_in_bytes, height, hipMemcpyDeviceToHost));
		
			CUT_SAFE_CALL( cutStopTimer( timer));
		}
		
		float time = cutGetAverageTimerValue(timer);
		CUT_SAFE_CALL( cutDeleteTimer( timer));

		CUDA_SAFE_CALL(hipHostFree(h_img));		

		printf("%f ms/f\n",time);
		printf("%f fps\n",1/(time*1e-3));	

	} else {
		printf( "Running with results being written out.  No timing results calculated. Size = %dx%d\n", width, height  );

		read_images( argv );
		init_flow();

		calculate_derivatives( 3 ); 	
		check_results( 2 );

		temporal_smoothing();
		check_results( 3 );

		second_order_smoothing(); 
		check_results( 4 );

		calculate_velocity();
		check_results( 5 );
 
		smooth_velocity(); 
		check_results( 6 );
	}

	clean_up();
}






